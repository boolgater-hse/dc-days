#include <cstdio>
#include <hip/hip_runtime.h>
#include <ctime>
#include <sys/time.h>

#define GET_TIME(now) { struct timeval t; gettimeofday(&t, NULL); now = t.tv_sec + t.tv_usec/1000000.0; }

#define VERBOSE

void serialSolution(int *matrixDimSize, double **initialMatrixCopy, double *resultsVectorForCpu) {
    for (int i = 0; i < *matrixDimSize - 1; i++) {
        for (int j = i + 1; j < *matrixDimSize; j++) {
            for (int k = i + 1; k < *matrixDimSize + 1; k++) {
                initialMatrixCopy[j][k] =
                        ((-initialMatrixCopy[j][i] / initialMatrixCopy[i][i]) * initialMatrixCopy[i][k]) + initialMatrixCopy[j][k];
            }
            initialMatrixCopy[j][i] = 0;
        }
    }

    for (int i = *matrixDimSize - 1; i >= 0; i--) {
        for (int j = *matrixDimSize - 1; j >= i; j--) {
            if (j == i) {
                resultsVectorForCpu[i] = initialMatrixCopy[i][*matrixDimSize] / initialMatrixCopy[i][j];
            } else {
                initialMatrixCopy[i][*matrixDimSize] -= initialMatrixCopy[i][j] * resultsVectorForCpu[j];
            }
        }
    }
}

__global__ void transformToUpperTriangularLinearSystem(double **initialMatrix, int i) {
    int currentThreadInRowNum = threadIdx.x;
    int currentRowNum = blockIdx.x;

    if (currentThreadInRowNum > i && currentRowNum > i && initialMatrix[currentRowNum][i] != 0) {
        initialMatrix[currentRowNum][currentThreadInRowNum] =
                ((-initialMatrix[currentRowNum][i] / initialMatrix[i][i]) * initialMatrix[i][currentThreadInRowNum]) +
                initialMatrix[currentRowNum][currentThreadInRowNum];
    }
    __syncthreads();
    if (currentThreadInRowNum == i && currentRowNum > i && initialMatrix[currentRowNum][i] != 0) {
        initialMatrix[currentRowNum][currentThreadInRowNum] = 0;
    }
}

__global__ void calculateLinearSystemResults(int *matrixDimSize, double **initialMatrix, double *resultsVector) {
    int currentThreadInBlockNum = threadIdx.x;

    double tempResult = initialMatrix[currentThreadInBlockNum][*matrixDimSize];

    for (int j = *matrixDimSize - 1; j >= 0; j--) {
        if (currentThreadInBlockNum == j) {
            resultsVector[j] = tempResult / initialMatrix[currentThreadInBlockNum][currentThreadInBlockNum];
        }
        __syncthreads();
        if (currentThreadInBlockNum < j) {
            tempResult -= initialMatrix[currentThreadInBlockNum][j] * resultsVector[j];
        }
    }
}

int main() {
    double start, end;
    int *matrixDimSize;
    double **initialMatrix;
    double *resultsVector;
    double **initialMatrixCopy;
    double *resultsVectorForCpu;

    hipMallocManaged(&matrixDimSize, 1 * sizeof(int));
    *matrixDimSize = 1000;

    hipMallocManaged(&resultsVector, *matrixDimSize * sizeof(double));
    hipMallocManaged(&initialMatrix, *matrixDimSize * sizeof(double *));
    for (int i = 0; i < *matrixDimSize; i++) {
        hipMallocManaged(&initialMatrix[i], (*matrixDimSize + 1) * sizeof(double));
    }
    resultsVectorForCpu = (double *) malloc(*matrixDimSize * sizeof(double));
    initialMatrixCopy = (double **) malloc(*matrixDimSize * sizeof(double *));
    for (int i = 0; i < *matrixDimSize; i++) {
        initialMatrixCopy[i] = (double *) malloc((*matrixDimSize + 1) * sizeof(double));
    }

    srand(time(NULL));
    for (int i = 0; i < *matrixDimSize; i++) {
        for (int j = 0; j < *matrixDimSize + 1; j++) {
            initialMatrix[i][j] = (double) rand();
            initialMatrixCopy[i][j] = initialMatrix[i][j];
        }
    }

    GET_TIME(start);
    serialSolution(matrixDimSize, initialMatrixCopy, resultsVectorForCpu);
    GET_TIME(end);

    printf("Computing time for single threaded CPU solution: %.9lf\n", end - start);
#ifdef VERBOSE
    printf("Solution:");
    for (int i = 0; i < *matrixDimSize; i++) {
        printf(" %lf", resultsVectorForCpu[i]);
    }
    printf("\n");
#endif

    GET_TIME(start);
    for (int i = 0; i < *matrixDimSize - 1; i++) {
        transformToUpperTriangularLinearSystem<<<*matrixDimSize, (*matrixDimSize + 1)>>>(initialMatrix, i);
    }
    calculateLinearSystemResults<<<1, *matrixDimSize>>>(matrixDimSize, initialMatrix, resultsVector);
    hipDeviceSynchronize();
    GET_TIME(end);

    printf("Computing time for CUDA solution: %.9lf\n", end - start);
#ifdef VERBOSE
    printf("Solution:");
    for (int i = 0; i < *matrixDimSize; i++) {
        printf(" %lf", resultsVector[i]);
    }
    printf("\n");
#endif

    for (int i = 0; i < *matrixDimSize; i++) {
        hipFree(initialMatrix[i]);
    }
    hipFree(initialMatrix);
    hipFree(resultsVector);
    for (int i = 0; i < *matrixDimSize; i++) {
        free(initialMatrixCopy[i]);
    }
    free(initialMatrixCopy);
    free(resultsVectorForCpu);
    hipFree(matrixDimSize);

    return 0;
}
